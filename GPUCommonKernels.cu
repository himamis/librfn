#include "hip/hip_runtime.h"
#include "GPUCommonKernels.h"

__global__ void setup_rng(hiprandState* rng_state, unsigned long seed) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &rng_state[tid]);
}

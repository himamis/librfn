#include "hip/hip_runtime.h"
/*
Copyright © 2015 Thomas Unterthiner
Licensed under GPL, version 2 or a later (see LICENSE.txt)
*/

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

#include "GPUDenseOperations.h"
#include "GPUDenseKernels.h"


GPUDenseOperations::GPUDenseOperations(const int n, const int m, const int k,
                                 unsigned long seed, int gpu_id) {

    // if no GPU was specified, try to pick the best one automatically
    if (gpu_id < 0) {
        gpu_id = get_gpu_id();
    }
    assert(gpu_id >= 0);
    hipSetDevice(gpu_id);

    // the following call does not work if the current process has already
    // called into librfn previously. Then, this call will return
    // hipErrorSetOnActiveProcess. Resetting the device won't work either,
    // because then the subsequent hipblasCreate call will just fail with
    // HIPBLAS_STATUS_NOT_INITIALIZED. I don't know why any of this is happening
    //CUDA_CALL(hipSetDeviceFlags(hipDeviceScheduleYield));

    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        const char* errmsg = cublasErrorString(status);
        fprintf(stderr, "CUBLAS initialization error: %s\n", errmsg);
        hipDeviceReset();
        throw std::runtime_error(errmsg);
    }
    CUSOLVER_CALL(hipsolverDnCreate(&cudense_handle));
    CUDA_CALL(hipMalloc(&rng_state, RNG_BLOCKS*RNG_THREADS*sizeof(hiprandState)));
    setup_rng<<<RNG_BLOCKS, RNG_THREADS>>>(rng_state, seed);
    int ones_size = n > k ? n : k;
    ones = malloc(ones_size*sizeof(float));
    fill(ones, ones_size, 1.0f);
    CUDA_CALL(hipMalloc(&devinfo, sizeof(int)));
}


GPUDenseOperations::~GPUDenseOperations() {
    free(devinfo);
    free(ones);
    for (auto i : buffer_map) {
        free(i.second);
    }
    CUSOLVER_CALL(hipsolverDnDestroy(cudense_handle));
    CUBLAS_CALL(hipblasDestroy(handle));
}


float* GPUDenseOperations::to_device(const float* src, size_t size) const {
    float* dst = 0;
    CUDA_CALL(hipMalloc(&dst, size));
    CUDA_CALL(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
    return dst;
}


void GPUDenseOperations::fill(float* X, const unsigned size, const float value) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    fill_eltw<<<blocks, threads>>>(X, size, value);
    assert(!hipGetLastError());
}


void GPUDenseOperations::dropout(float* X, const unsigned size,
                                    const float dropout_rate) const {
    dropout_eltw<<<RNG_BLOCKS, RNG_THREADS>>>(X, size, dropout_rate, rng_state);
    assert(!hipGetLastError());
}


void GPUDenseOperations::add_gauss_noise(float* X, const unsigned size,
                                    const float noise_rate) const {
    gauss_noise_eltw<<<RNG_BLOCKS, RNG_THREADS>>>(X, size, noise_rate, rng_state);
    assert(!hipGetLastError());
}


void GPUDenseOperations::add_saltpepper_noise(float* X, const unsigned size,
                                    const float noise_rate) const {
    saltpepper_noise_eltw<<<RNG_BLOCKS, RNG_THREADS>>>(X, size, noise_rate, rng_state);
    assert(!hipGetLastError());
}


void GPUDenseOperations::invert(float* X, const unsigned size) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    invert_eltw<<<blocks, threads>>>(X, size);
    assert(!hipGetLastError());
}


void GPUDenseOperations::maximum(float* x, const float value, const unsigned size) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    maximum_eltw<<<blocks, threads>>>(x, value, size);
    assert(!hipGetLastError());
}


void GPUDenseOperations::leaky_relu(float* x, const float value, const unsigned size) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    leaky_relu_eltw<<<blocks, threads>>>(x, value, size);
    assert(!hipGetLastError());
}


void GPUDenseOperations::sigmoid(float* x, const unsigned size) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    sigmoid_eltw<<<blocks, threads>>>(x, size);
    assert(!hipGetLastError());
}


void GPUDenseOperations::tanh(float* x, const unsigned size) const {
    int threads, blocks;
    get_grid_sizes(size, &threads, &blocks);
    tanh_eltw<<<blocks, threads>>>(x, size);
    assert(!hipGetLastError());
}


void GPUDenseOperations::soft_threshold(float* x, const float alpha, const unsigned size) const {
   int threads, blocks;
   get_grid_sizes(size, &threads, &blocks);
   softthreshold_eltw<<<blocks, threads>>>(x, alpha, size);
   assert(!hipGetLastError());
}



void GPUDenseOperations::fill_eye(float* X, unsigned n) const {
    memset(X, 0, n*n*sizeof(float));
    axpy(n, 1.0f, ones, 0, X, n+1);
}


void GPUDenseOperations::calculate_column_variance(float* X, const unsigned nrows,
                                               const unsigned ncols, float* variance) const {
    int threads, blocks;
    get_grid_sizes(ncols, &threads, &blocks);
    col_variance_kernel<<<threads, blocks>>>(X, variance, nrows, ncols);
}


void GPUDenseOperations::invsqrt(float* s, const unsigned n) const {
    int t, b;
    get_grid_sizes(n, &t, &b);
    invsqrt_eltw<<<t, b>>>(s, n);
}

void GPUDenseOperations::scale_columns(float* X, const unsigned nrows, const unsigned ncols, float* s) const {

    int threads, blocks;
    get_grid_sizes(ncols*nrows, &threads, &blocks);
    scale_columns_kernel<<<threads, blocks>>>(X, s, nrows, ncols);
}


void GPUDenseOperations::scale_rows(float* X, const unsigned nrows, const unsigned ncols, float* s) const {
    int threads, blocks;
    get_grid_sizes(ncols*nrows, &threads, &blocks);
    scale_rows_kernel<<<threads, blocks>>>(X, s, nrows, ncols);
}


void GPUDenseOperations::printMatrixRM(const float* a, int n, int m, const char* fmt) {
    const char* format = fmt == 0 ? "%1.3f " : fmt;
    size_t size = n*m*sizeof(float);
    float* tmp = (float*) std::malloc(size);
    CUDA_CALL(hipMemcpy(tmp, a, size, hipMemcpyDeviceToHost));
	for (int i = 0; i < n; ++i) {
		for (int j =0 ; j < m; ++j)
			printf(format, tmp[i*m + j]);
		printf("\n");
	}
    printf("\n");
    std::free(tmp);
}


void GPUDenseOperations::printMatrixCM(const float* a, int n, int m, const char* fmt) {
    const char* format = fmt == 0 ? "%1.3f " : fmt;
    size_t size = n*m*sizeof(float);
    float* tmp = (float*) std::malloc(size);
    CUDA_CALL(hipMemcpy(tmp, a, size, hipMemcpyDeviceToHost));
	for (int i = 0; i < n; ++i) {
		for (int j =0 ; j < m; ++j)
			printf(format, tmp[i + j*n]);
		printf("\n");
	}
    printf("\n");
    std::free(tmp);
}

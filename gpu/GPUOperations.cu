#include "hip/hip_runtime.h"
#include "GPUOperations.h"
#include "GPUCommonKernels.h"
#include "SparseMatrix.h"

template<typename MatrixType>
GPUOperations<MatrixType>::GPUOperations(const int n, const int m, const int k, unsigned long seed, int gpu_id) {
	// if no GPU was specified, try to pick the best one automatically
	if (gpu_id < 0) {
		gpu_id = get_gpu_id();
	}
	assert(gpu_id >= 0);
	hipSetDevice (gpu_id);

	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		const char* errmsg = cublasErrorString(status);
		fprintf(stderr, "CUBLAS initialization error: %s\n", errmsg);
		hipDeviceReset();
		throw std::runtime_error(errmsg);
	}
	CUSOLVER_CALL(hipsolverDnCreate(&cudense_handle));
	CUDA_CALL(hipMalloc(&devinfo, sizeof(int)));
	CUDA_CALL(hipMalloc(&rng_state, RNG_BLOCKS * RNG_THREADS * sizeof(hiprandState)));
	setup_rng<<<RNG_BLOCKS, RNG_THREADS>>>(rng_state, seed);
	int ones_size = n > k ? n : k;
	ones = malloc(ones_size * sizeof(float));
	fill(ones, ones_size, 1.0f);
}

template<typename MatrixType>
GPUOperations<MatrixType>::~GPUOperations() {
	free(devinfo);

	for (auto i : buffer_map) {
		free(i.second);
	}
	CUSOLVER_CALL(hipsolverDnDestroy(cudense_handle));
	CUBLAS_CALL(hipblasDestroy(handle));
}

template<typename MatrixType>
float* GPUOperations<MatrixType>::to_device(const float* src, size_t size) const {
	float* dst = 0;
	CUDA_CALL(hipMalloc(&dst, size));
	CUDA_CALL(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
	return dst;
}

template<typename MatrixType>
void* GPUOperations<MatrixType>::memset(void* dest, int ch, size_t count) const {
	CUDA_CALL(hipMemset(dest, ch, count));
	return dest;
}

template<typename MatrixType>
float* GPUOperations<MatrixType>::memcpy(void* dest, const void *src, size_t count) const {
	CUDA_CALL(hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice));
	return 0;
}

template<typename MatrixType>
void GPUOperations<MatrixType>::free(void* ptr) const {
	if (ptr != 0)
		CUDA_CALL(hipFree(ptr));
}

template<typename MatrixType>
void GPUOperations<MatrixType>::free_devicememory(void* ptr) const {
	if (ptr != 0)
		CUDA_CALL(hipFree(ptr));
}

template<typename MatrixType>
float* GPUOperations<MatrixType>::malloc(size_t size) const {
	float* retval = 0;
	hipError_t err = hipMalloc(&retval, size);
	CUDA_CALL(err);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMalloc failed\n");
		retval = 0;
	}
	return retval;
}

template<typename MatrixType>
void GPUOperations<MatrixType>::fill(float* X, const unsigned size, const float value) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	fill_eltw<<<blocks, threads>>>(X, size, value);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::invert(float* X, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	invert_eltw<<<blocks, threads>>>(X, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::maximum(float* x, const float value, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	maximum_eltw<<<blocks, threads>>>(x, value, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::leaky_relu(float* x, const float value, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	leaky_relu_eltw<<<blocks, threads>>>(x, value, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::sigmoid(float* x, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	sigmoid_eltw<<<blocks, threads>>>(x, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::tanh(float* x, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	tanh_eltw<<<blocks, threads>>>(x, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::soft_threshold(float* x, const float alpha, const unsigned size) const {
	int threads, blocks;
	get_grid_sizes(size, &threads, &blocks);
	softthreshold_eltw<<<blocks, threads>>>(x, alpha, size);
	assert(!hipGetLastError());
}

template<typename MatrixType>
void GPUOperations<MatrixType>::fill_eye(float* X, unsigned n) const {
	memset(X, 0, n * n * sizeof(float));
	axpy(n, 1.0f, ones, 0, X, n + 1);
}

template<typename MatrixType>
void GPUOperations<MatrixType>::invsqrt(float* s, const unsigned n) const {
	int t, b;
	get_grid_sizes(n, &t, &b);
	invsqrt_eltw<<<t, b>>>(s, n);
}

// force compiler to create related class
template class GPUOperations<float*>;
template class GPUOperations<sparse_matrix_csr>;

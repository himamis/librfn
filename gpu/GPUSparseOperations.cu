#include "GPUSparseOperations.h"
#include "GPUSparseKernels.h"

GPUSparseOperations::GPUSparseOperations(const int n, const int m, const int k, unsigned long seed, int gpu_id) :
		GPUOperations(n, m, k, seed, gpu_id) {
	hipsparseStatus_t status = hipsparseCreate(&sparseHandle);

	if (status != HIPSPARSE_STATUS_SUCCESS) {
		fprintf(stderr, "cuSparse: %d\n", status);
		hipDeviceReset();
		throw std::runtime_error("cuSparse error");
	}
}

GPUSparseOperations::~GPUSparseOperations() {
	CUSPARSE_CALL(hipsparseDestroy(sparseHandle));
}

void GPUSparseOperations::fill_eye(hipsparseMatDescr_t, unsigned int n) const {

}

void GPUSparseOperations::gemm(const char *transa, const char *transb, const int m, const int n, const int k,
		const float alpha, const float *a, const int lda, const float *b, const int ldb, const float beta, float *c,
		const int ldc) const {
	//CUSPARSE_CALL(hipsparseSgemmi())

}

void GPUSparseOperations::scale_rows(hipsparseMatDescr_t, const unsigned nrows, const unsigned ncols, float* s) const {
	int threads, blocks;
	get_grid_sizes(ncols * nrows, &threads, &blocks);
}

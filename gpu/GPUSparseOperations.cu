#include "GPUSparseOperations.h"
#include "GPUSparseKernels.h"

GPUSparseOperations::GPUSparseOperations(const int n, const int m, const int k, unsigned long seed, int gpu_id) :
		GPUOperations(n, m, k, seed, gpu_id) {
	hipsparseStatus_t status = hipsparseCreate(&sparseHandle);

	if (status != HIPSPARSE_STATUS_SUCCESS) {
		fprintf(stderr, "cuSparse: %d\n", status);
		hipDeviceReset();
		throw std::runtime_error("cuSparse error");
	}
}

GPUSparseOperations::~GPUSparseOperations() {
	CUSPARSE_CALL(hipsparseDestroy(sparseHandle));
}

void GPUSparseOperations::calculate_column_variance(hipsparseMatDescr_t X, const unsigned nrows, const unsigned ncols,
			float* variances) const {

}

void GPUSparseOperations::scale_columns(hipsparseMatDescr_t X, const unsigned nrows, const unsigned ncols, float* s) const {

}

void GPUSparseOperations::scale_rows(hipsparseMatDescr_t X, const unsigned nrows, const unsigned ncols, float* s) const {

}

void GPUSparseOperations::dropout(hipsparseMatDescr_t X, const unsigned size, const float dropout_rate) const {

}

void GPUSparseOperations::add_saltpepper_noise(hipsparseMatDescr_t X, const unsigned size, const float noise_rate) const {

}

void GPUSparseOperations::add_gauss_noise(hipsparseMatDescr_t X, const unsigned size, const float noise_rate) const {

}

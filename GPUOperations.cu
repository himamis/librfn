#include "GPUOperations.h"
#include "GPUCommon.h"

template<typename MatrixType>
void* GPUOperations<MatrixType>::memset(void* dest, int ch, size_t count) const {
	CUDA_CALL(hipMemset(dest, ch, count));
	return dest;
}

template<typename MatrixType>
float* GPUOperations<MatrixType>::memcpy(void* dest, const void *src, size_t count) const {
	CUDA_CALL(hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice));
	return 0;
}

template<typename MatrixType>
void GPUOperations<MatrixType>::free(void* ptr) const {
	if (ptr != 0)
		CUDA_CALL(hipFree(ptr));
}

template<typename MatrixType>
void GPUOperations<MatrixType>::free_devicememory(void* ptr) const {
	if (ptr != 0)
		CUDA_CALL(hipFree(ptr));
}

template<typename MatrixType>
float* GPUOperations<MatrixType>::malloc(size_t size) const {
	float* retval = 0;
	hipError_t err = hipMalloc(&retval, size);
	CUDA_CALL(err);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMalloc failed\n");
		retval = 0;
	}
	return retval;
}

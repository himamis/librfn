#include "hip/hip_runtime.h"
#include "GPUDenseKernels.h"

__global__ void dropout_eltw(float* x, const unsigned size, const float dropout_rate, hiprandState* rng_state) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	hiprandState localState = rng_state[tid];
	for (unsigned i = tid; i < size; i += num_threads)
		x[i] = (hiprand_uniform(&localState) < dropout_rate) ? 0.0 : x[i];
	rng_state[tid] = localState;
}

__global__ void saltpepper_noise_eltw(float* x, const unsigned size, const float noise_rate, hiprandState* rng_state) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	hiprandState localState = rng_state[tid];
	for (unsigned i = tid; i < size; i += num_threads)
		if (hiprand_uniform(&localState) < noise_rate) {
			x[i] = (hiprand_uniform(&localState) < 0.5f) ? 0.0f : 1.0f;
		}
	rng_state[tid] = localState;

}

__global__ void gauss_noise_eltw(float* x, const unsigned size, const float noise_rate, hiprandState* rng_state) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	hiprandState localState = rng_state[tid];
	for (unsigned i = tid; i < size; i += num_threads)
		x[i] += hiprand_normal(&localState) * noise_rate;
	rng_state[tid] = localState;

}

__global__ void leaky_relu_eltw(float* x, const float value, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = (x[i] < 0.0f) ? x[i] * value : x[i];
	}
}

__global__ void maximum_eltw(float* x, const float value, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = fmaxf(x[i], value);
	}
}

__global__ void sigmoid_eltw(float* x, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = 1 / (1 + __expf(-x[i]));
	}
}

__global__ void tanh_eltw(float* x, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = tanhf(x[i]);
	}
}

__global__ void softthreshold_eltw(float* x, float alpha, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		const float f = x[i];
		x[i] = f > 0 ? fmaxf(0., f - alpha) : fminf(0., f + alpha);
	}
}

__global__ void fill_eltw(float* x, const unsigned size, const float value) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = value;
	}
}

__global__ void invert_eltw(float* x, const unsigned size) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = gridDim.x * blockDim.x;
	for (unsigned i = tid; i < size; i += num_threads) {
		x[i] = 1.0f / x[i];
	}
}

__global__ void col_variance_kernel(const float* X, float* var, const unsigned nrows, const unsigned ncols) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = blockDim.x * gridDim.x;
	for (unsigned i = tid; i < ncols; i += num_threads) {
		var[i] = 0.0;
		for (unsigned j = 0; j < nrows; ++j) {
			var[i] += X[j * ncols + i];
		}
		float m = var[i] / nrows;
		var[i] = 0.0;
		for (unsigned j = 0; j < nrows; ++j) {
			float tmp = X[j * ncols + i] - m;
			var[i] += tmp * tmp;
		}
		var[i] /= nrows;
	}
}

__global__ void invsqrt_eltw(float* x, const unsigned k) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = blockDim.x * gridDim.x;
	for (unsigned i = tid; i < k; i += num_threads) {
		x[i] = (x[i] > 1e-7) ? rsqrtf(x[i]) : 1.0;
	}
}

__global__ void scale_columns_kernel(float* X, float* a, const unsigned nrows, const unsigned ncols) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = blockDim.x * gridDim.x;
	for (unsigned i = tid; i < ncols * nrows; i += num_threads) {
		X[i] *= a[i % ncols];
	}
}

__global__ void scale_rows_kernel(float* X, float* a, const unsigned nrows, const unsigned ncols) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned num_threads = blockDim.x * gridDim.x;
	for (unsigned i = tid; i < ncols * nrows; i += num_threads) {
		X[i] *= a[i / ncols];
	}
}
